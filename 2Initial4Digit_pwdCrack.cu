#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
 This is a program of password cracking of two alphabets and four digits using cuda. 

 
 Compile BY:
    nvcc -o 2Initial4Digit_pwdCrack 2Initial4Digit_pwdCrack.cu 
  
  Run BY:
    ./2Initial4Digit_pwdCrack >2Initial4Digit_Password
 
  To calculate mean time execute the program 10 times and grep time in second only csv file:
   ./mr.py ./2Initial4Digit_pwdCrack | grep Time | awk -F ' ' '{print $6}' | sed 's/s//'> BgnKndl_MeanTime_2Initial4Digit_pwdCrack.csv
  
   =============================================================
        Name: BIGYAN KANDEL  University ID: "1917852"
   =============================================================
********************************************************************************/


__device__ void displayResult(char *password)
{
 printf("Matching password is: %s\n",password);
}

__device__ void is_a_match(char *attempt) {
  char plain_password_1[] = "BI1996";
  char plain_password_2[] = "GY2053";
  char plain_password_3[] = "AN9867";
  char plain_password_4[] = "BK9658";

//send values from kernal as attempt
  char *a = attempt;
  char *b = attempt;
  char *c = attempt;
  char *d = attempt;
  char *p1 = plain_password_1;
  char *p2 = plain_password_2;
  char *p3 = plain_password_3;
  char *p4 = plain_password_4;

//Checking each alphabets of password1 with attempt as a 
  while(*a == *p1) { 
   if(*a == '\0') 
    {
        displayResult(plain_password_1);
      break;
    }

    a++;
    p1++;
  }
	//Checking each alphabets of password2 with attempt as b
  while(*b == *p2) { 
   if(*b == '\0') 
    {
        displayResult(plain_password_2);
      break;
    }

    b++;
    p2++;
  }

//Checking each alphabets of password3 with attempt as c
  while(*c == *p3) { 
   if(*c == '\0') 
    {
	displayResult(plain_password_3);
      break;
    }

    c++;
    p3++;
  }

//Checking each alphabets of password4 with attempt as d
  while(*d == *p4) { 
   if(*d == '\0') 
    {
       displayResult(plain_password_4);
      break;
    }

    d++;
    p4++;
  }
  return;

}


__global__ void  kernel() {
char w,x,y,z;
  
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstAlp = i; 
char secondAlp = j; 
    
password[0] = firstAlp;
password[1] = secondAlp;
  for(w='0'; w<='9'; w++){
   for(x='0'; x<='9'; x++){
    for(y='0'; y<='9'; y++){
     for(z='0'; z<='9'; z++){
	password[2] = w;
	password[3] = x;
	password[4] = y;
	password[5] = z; 
      is_a_match(password); // calling device function passing each password
     }
    }
   }
  }

}

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);
  
  dim3 block_Dim(26,1,1), thread_Dim(26,1,1);        
  kernel <<<block_Dim,thread_Dim>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}


